#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 11.79

#define WIDTH 10
#define TILE_WIDTH 10

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float* ad, float* bd, float* cd, int width);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
	}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i];
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	int size = n * sizeof(float);

	hipMalloc((void**) &ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &bd, size);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	
	hipMalloc((void**) &cd, size);

	dim3 dimGrid(WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	// Kernal invocation
	vecGPU<<<1, n>>>(ad, bd, cd, n);

	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
	hipFree(ad); 
	hipFree(bd); 
	hipFree(cd);

	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if(temp[i] != c[i])
		printf("Element %d in the result array does not match the sequential version (%lf vs. %lf)\n", i, c[i], temp[i]);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float* ad, float* bd, float* cd, int width) {
	// int index = blockIdx.x * width;
	int index = blockIdx.x;

	float c_value = 0;

	for(int j = 0; j < width; j++) {
		// if(index + j < width) {
		// 	c_value += ad[index + j] * bd[index + j];
		// }
		c_value += ad[j] * bd[j];
	}

	cd[index] = c_value;
}