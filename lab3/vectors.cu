#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 11.79

#define BLOCKS_PER_GRID 4
#define THREADS_PER_BLOCK 500

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float* ad, float* bd, float* cd, int width);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
	}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i];
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	int size = n * sizeof(float);

	hipMalloc((void**) &ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &bd, size);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &cd, size);
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
	
	// dim3 dimGrid(n / TILE_WIDTH);
	// dim3 dimBlock(TILE_WIDTH);

	// Kernal invocation
	vecGPU<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(ad, bd, cd, n);

	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
	hipFree(ad); 
	hipFree(bd); 
	hipFree(cd);

	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++) {
	  if(temp[i] - c[i] > 0.001 || temp[i] - c[i] < -0.001 )
		printf("Element %d in the result array does not match the sequential version (%lf vs. %lf)\n", i, c[i], temp[i]);
	//   if(temp[i] != c[i])
	// 	printf("Element %d in the result array does not match the sequential version (%lf vs. %lf)\n", i, c[i], temp[i]);
	}
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float* ad, float* bd, float* cd, int width) {
	int additional_work = 0;
	int calcs_per_thead = width / (BLOCKS_PER_GRID * THREADS_PER_BLOCK);
	int index = (blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) * calcs_per_thead;

	if(blockIdx.x == BLOCKS_PER_GRID - 1) {
		// We are in the last block
		if(width > BLOCKS_PER_GRID * THREADS_PER_BLOCK * calcs_per_thead) {
			additional_work = width - (BLOCKS_PER_GRID * THREADS_PER_BLOCK * calcs_per_thead);

			if(threadIdx.x < additional_work) {
				printf("______________________SDFSDFSDFD______________________\n");
				cd[index + THREADS_PER_BLOCK*calcs_per_thead + threadIdx.x] += ad[index + THREADS_PER_BLOCK*calcs_per_thead + threadIdx.x] * bd[index + THREADS_PER_BLOCK*calcs_per_thead + threadIdx.x];
			}
		}
	}

	for(int j = 0; j < calcs_per_thead; j++) {
		cd[index + j] += ad[index + j] * bd[index + j];
	}

	// if(index < width)
	// cd[index] += ad[index] * bd[index];
}

/**

1,000,000 width


4 blocks
500 threads per block

2000 threads in total

Each thread needs to do 500 calculations



2002 entries
4 blocks
500 threads per block
2000 threads in total

Each thread needs to do 1 calculation 

BUT there are two extra entries at the end, when we are in the last block we need to recognize
that and assign that work to two threads.


if in last block:
	if n > num_blocks*threads_per_block*work_per_thread:
		additional_work = n % num_blocks*threads_per_block*work_per_thread; // in this case this will be 2


for (...)
	if j < additional_work:
		// We behave as if there is an additional thread.
		cd[index + calcs_per_thead + j] += ad[index + calcs_per_thead + j] * bd[index + calcs_per_thead + j];




*/