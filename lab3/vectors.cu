#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 11.79

#define BLOCKS_PER_GRID 16
#define THREADS_PER_BLOCK 250

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float* ad, float* bd, float* cd, int calcs_per_thead, int width);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
	}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i];
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	printf("Total time taken by the sequential part = %d\n", (end - start)*1000);
    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	int size = n * sizeof(float);

	hipMalloc((void**) &ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &bd, size);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &cd, size);
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

	int calcs_per_thread = n / (BLOCKS_PER_GRID * THREADS_PER_BLOCK);

	// Kernal invocation
	vecGPU<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(ad, bd, cd, calcs_per_thread, n);

	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
	hipFree(ad); 
	hipFree(bd); 
	hipFree(cd);

	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	// for(i = 0; i < n; i++) {
	//   if(temp[i] != c[i])
	// 	printf("Element %d in the result array does not match the sequential version (%lf vs. %lf)\n", i, c[i], temp[i]);
	// }
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}

/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float* ad, float* bd, float* cd, int calcs_per_thread, int width) {
	int global_id = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	int index = global_id * calcs_per_thread;
	int base_work = BLOCKS_PER_GRID * THREADS_PER_BLOCK * calcs_per_thread;
	int additional_work = width - base_work;

	for(int j = 0; j < calcs_per_thread; j++) {
		cd[index + j] += ad[index + j] * bd[index + j];
	}

	if(global_id < additional_work) {
		cd[base_work + global_id] += ad[base_work + global_id] * bd[base_work + global_id];
	}
}
